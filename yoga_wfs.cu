#include "hip/hip_runtime.h"
#include <yoga_wfs.h>
bool isPow2(unsigned int x)
{
    return ((x&(x-1))==0);
}

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template<class T> struct SharedMemory
{
    __device__ inline operator       T*()
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }

    __device__ inline operator const T*() const
    {
        extern __shared__ int __smem[];
        return (T*)__smem;
    }
};

// specialize for double to avoid unaligned memory 
// access compile errors
template<> struct SharedMemory<double>
{
    __device__ inline operator       double*()
    {
        extern __shared__ double __smem_d[];
        return (double*)__smem_d;
    }

    __device__ inline operator const double*() const
    {
        extern __shared__ double __smem_d[];
        return (double*)__smem_d;
    }
};



__global__ void fillcamplipup_krnl(hipFloatComplex *amplipup, float *phase,float *offset, float *mask, int *indx, int Nfft, int Npup, int npup, int N)
{
  int nim,idim,idimx,idimy,idx;
  int tid   = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    nim   = tid / Npup;
    idim  = tid - nim * Npup;
    idimx = idim % npup;
    idimy = idim / npup;
    idx   = idimx + idimy * Nfft + nim * Nfft * Nfft;
    amplipup[idx].x = (cosf(phase[indx[tid]]-offset[idim]))*mask[indx[tid]];
    amplipup[idx].y = (sinf(phase[indx[tid]]-offset[idim]))*mask[indx[tid]];
    //amplipup[idx].x = mask[indx[tid]] * cosf(-offset[idim]);
    //amplipup[idx].y = mask[indx[tid]] * sinf(-offset[idim]);
    tid  += blockDim.x * gridDim.x;
  }
}

int fillcamplipup(hipFloatComplex *amplipup, float *phase, float *offset, float *mask, int *indx, int Nfft, int Npup, int Nsub, int npup, int device)
// here amplipup is a cube of data of size nfft x nfft x nsubap
// phase is an array of size pupdiam x pupdiam
// offset is an array of size pdiam x pdiam
// mask is an array of size pupdiam x pupdiam
// indx is an array of size pdiam x pdiam x nsubap
// number of thread required : pdiam x pdiam x nsubap
// Npup = pdiam x pdiam
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
  
  int N = Npup * Nsub;

  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  fillcamplipup_krnl<<<grid, threads>>>(amplipup,phase,offset,mask,indx,Nfft,Npup,npup,N);
  cutilCheckMsg("fillcamplipup_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}



__global__ void fillbincube_krnl(float *bcube, float *hrimage, int *indxpix, int Nfft, int Npix, int Nrebin, int N)
{
  /*
    indx is an array nrebin^2 * npix^2
    it gives the nrebin x nrebin pixels in the hrimage per npix x npix pixels of the subap
    Npix = npix x npix
   */
  int npix,nsubap,nrebin;
  int tid     = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    nsubap  = tid / Npix;
    npix    = tid % Npix;
    for (int i=0;i<Nrebin;i++) {
      nrebin = indxpix[i+npix*Nrebin];
      bcube[tid] += hrimage[nrebin + Nfft*nsubap];
    }
    tid  += blockDim.x * gridDim.x;
  }
}

int fillbincube(float *bcube, float *hrimage, int *indxpix, int Nfft, int Npix, int Nrebin, int Nsub, int device)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
  
  int N = Npix * Nsub;
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  fillbincube_krnl<<<grid, threads>>>(bcube,hrimage,indxpix,Nfft,Npix,Nrebin,N);
  cutilCheckMsg("fillbincube_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}



__global__ void fillbinimg_krnl(float *bimage, float *bcube, int *indxbin, int N)
{
  /*
    indx is an array nrebin^2 * npix^2
    it gives the nrebin x nrebin pixels in the hrimage per npix x npix pixels of the subap
    Npix = npix x npix
   */
  int tid     = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    bimage[indxbin[tid]] = bcube[tid];
    tid  += blockDim.x * gridDim.x;
  }
}

__global__ void addbinimg_krnl(float *bimage, float *bcube, int *indxbin, int N)
{
  /*
    indx is an array nrebin^2 * npix^2
    it gives the nrebin x nrebin pixels in the hrimage per npix x npix pixels of the subap
    Npix = npix x npix
   */
  int tid     = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    bimage[indxbin[tid]] += bcube[tid];
    tid  += blockDim.x * gridDim.x;
  }
}

int fillbinimg(float *bimage, float *bcube, int *indxbin, int Npix, int Nsub, bool add, int device)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
  
  int N = Npix * Nsub;
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  if (add)
    addbinimg_krnl<<<grid, threads>>>(bimage,bcube,indxbin,N);
  else 
    fillbinimg_krnl<<<grid, threads>>>(bimage,bcube,indxbin,N);

  cutilCheckMsg("fillbinimg_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}



template<class T> __global__ void indexfill_krnl(T *odata, T *idata, int *indx,int ntot, int Ntot, int N)
{
  int nim, idim;
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    nim = tid / ntot;
    idim = tid - (nim * ntot);
    odata[indx[idim] + (nim * Ntot)] = idata[tid];
    tid += blockDim.x * gridDim.x;
  }
}

int indexfill(float *d_odata,float *d_idata,int *indx,int nx, int Nx, int N,int device)
{

  int ntot = nx * nx;
  int Ntot = Nx * Nx;
  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  indexfill_krnl<<<grid, threads>>>(d_odata, d_idata,indx, ntot, Ntot, N);

  return EXIT_SUCCESS;
}



__global__ void cfillrealp_krnl(hipFloatComplex *odata, float *idata, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[tid].x = idata[tid];
    tid += blockDim.x * gridDim.x;
  }
}

int cfillrealp(hipFloatComplex *d_odata,float *d_idata,int N,int device)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  cfillrealp_krnl<<<grid, threads>>>(d_odata, d_idata, N);

   return EXIT_SUCCESS;
}



__global__ void conv_krnl(hipFloatComplex *odata,hipFloatComplex *idata, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hipFloatComplex tmp;

  while (tid < N) {
    tmp.x = idata[tid].x*odata[tid].x-idata[tid].y*odata[tid].y;
    tmp.y = idata[tid].y*odata[tid].x+idata[tid].x*odata[tid].y;
    odata[tid] = tmp;
    tid += blockDim.x * gridDim.x;
  }
}

int convolve(hipFloatComplex *d_odata,hipFloatComplex *d_idata,int N,int device)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  conv_krnl<<<grid, threads>>>(d_odata, d_idata, N);

   return EXIT_SUCCESS;
}



__global__ void cgetrealp_krnl(float *odata, hipFloatComplex *idata, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[tid] = idata[tid].x;
    tid += blockDim.x * gridDim.x;
  }
}

int cgetrealp(float *d_odata,hipFloatComplex *d_idata,int N,int device)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  cgetrealp_krnl<<<grid, threads>>>(d_odata, d_idata, N);

   return EXIT_SUCCESS;
}



/*
    This version uses sequential addressing -- no divergence or bank conflicts.
*/
template <class T> __device__ void reduce_krnl(T *sdata, int size, int n)
{
  if (!((size&(size-1))==0)) {
    unsigned int s;
    if (size %2 != 0) s = size/2+1;
    else s = size/2;
    unsigned int s_old = size;
    while (s>0) {
      if ((n < s) && (n + s < s_old)) {
	sdata[n] += sdata[n + s];
      }
      __syncthreads();
      s_old = s;
      s /= 2;
      if ((2*s < s_old) && (s!=0)) s += 1;
    }
  } else {
    // do reduction in shared mem
    for(unsigned int s=size/2; s>0; s>>=1) {	
      if (n < s) {
	sdata[n] += sdata[n + s];
      }
      __syncthreads();
    }
  }
}


template <class T> __device__ void scanmax_krnl(T *sdata, int *values,int size, int n)
{
  if (!((size&(size-1))==0)) {
    unsigned int s;
    if (size %2 != 0) s = size/2+1;
    else s = size/2;
    unsigned int s_old = size;
    while (s>0) {
      if ((n < s) && (n + s < s_old)) {
	if (sdata[n] < sdata[n + s]) {
	  values[n] = n + s;
	  sdata[n] = sdata[n+s];
	}
      }
      __syncthreads();
      s_old = s;
      s /= 2;
      if ((2*s < s_old) && (s!=0)) s += 1;
    }
  } else {
    // do reduction in shared mem
    for(unsigned int s=size/2; s>0; s>>=1) {	
      if (n < s) {
	if (sdata[n] < sdata[n + s]) {
	  values[n] = n + s;
	  sdata[n] = sdata[n+s];
	}
      }
      __syncthreads();
    }
  }
}


template <class T> 
__device__ inline void mswap(T & a, T & b)
{
    T tmp = a;
    a = b;
    b = tmp;
}


template <class T> __device__ inline void sortmax_krnl(T *sdata, unsigned int *values,int size, int n)
{

  if (!((size&(size-1))==0)) {
    unsigned int s;
    if (size %2 != 0) s = size/2+1;
    else s = size/2;
    unsigned int s_old = size;
    while (s>0) {
      if ((n < s) && (n + s < s_old)) {
	if (sdata[n] < sdata[n + s]) {
	  mswap(values[n],values[n+s]);
	  mswap(sdata[n],sdata[n+s]);
	}
      }
      __syncthreads();
      s_old = s;
      s /= 2;
      if ((2*s < s_old) && (s!=0)) s += 1;
    }
  } else {
    // do reduction in shared mem
    for(unsigned int s=size/2; s>0; s>>=1) {	
      if (n < s) {
	if (sdata[n] < sdata[n + s]) {
	  mswap(values[n],values[n+s]);
	  mswap(sdata[n],sdata[n+s]);
	}
      }
      __syncthreads();
    }
  }
}

template <class T> __global__ void reduce2(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    sdata[tid] = (i < n) ? g_idata[i] : 0;
    
    __syncthreads();

    reduce_krnl(sdata,blockDim.x,tid);
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <class T> __global__ void reduce2(T *g_idata, T *g_odata, T thresh, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (i < n) sdata[tid] = (g_idata[i] > thresh) ? g_idata[i] : 0;
    
    __syncthreads();

    reduce_krnl(sdata,blockDim.x,tid);
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

template <class T> __global__ void sortmax(T *g_idata, T *g_odata, int *values, int nmax)
{
  extern __shared__ uint svalues[];
  T *sdata = (T*)&svalues[blockDim.x];
  
  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  svalues[tid] = tid;
  sdata[tid]   = g_idata[i];
  
  __syncthreads();
  
  for (int cc=0;cc<nmax;cc++) {
    
    if (tid >= cc) sortmax_krnl(&(sdata[cc]),&(svalues[cc]),blockDim.x-cc,tid-cc);
    
    __syncthreads();
    
  } 
  
  if (tid < nmax) {
    g_odata[nmax*blockIdx.x+tid] = sdata[tid];
    values[nmax*blockIdx.x+tid]  = svalues[tid];
  }
  __syncthreads();
 
}

template <class T> __global__ void centroid_max(T *g_idata, T *g_odata, int n, int nmax, int nsub)
{
  extern __shared__ uint svalues[];
  T *sdata = (T*)&svalues[blockDim.x];
  T subsum;

  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  svalues[tid] = tid;
  sdata[tid]   = g_idata[i];
  
  __syncthreads();
  
  for (int cc=0;cc<nmax;cc++) {
    
    if (tid >= cc) sortmax_krnl(&(sdata[cc]),&(svalues[cc]),blockDim.x-cc,tid-cc);
    
    __syncthreads();
    
  } 
  // at this point the nmax first elements of sdata are the nmax brightest
  // pixels and the nmax first elements of svalues are their positions

  // first copy the brightest values out for reduction  
  if ((tid >= nmax) && (tid < 2*nmax)) sdata[tid] = sdata[tid-nmax];

  __syncthreads();

  reduce_krnl(sdata,nmax,tid);

  // get the sum per subap  
  if (tid == 0) subsum = sdata[tid];

  // put back the brightest pixels values 
  if ((tid >= nmax) && (tid < 2*nmax)) sdata[tid-nmax] = sdata[tid];

  __syncthreads();

  // compute the centroid on the first part of the array
  if (tid < nmax) sdata[tid] *= ((svalues[tid] % n) + 1);
  // x centroid
  __syncthreads();
  reduce_krnl(sdata,nmax,tid);

  if (tid == 0) g_odata[blockIdx.x] = sdata[tid]/subsum;

  // put back the brightest pixels values 
  if ((tid >= nmax) && (tid < 2*nmax)) sdata[tid-nmax] = sdata[tid];

  __syncthreads();

  // compute the centroid on the first part of the array
  if (tid < nmax) sdata[tid] *= (svalues[tid] / n + 1);
  // y centroid
  __syncthreads();
  reduce_krnl(sdata,nmax,tid);

  if (tid == 0) g_odata[blockIdx.x+nsub] = sdata[tid]/subsum;
 
}


template <class T> void subap_reduce(int size, int threads, int blocks, T *d_idata, T *d_odata)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps 
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
    reduce2<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);

    cutilCheckMsg("reduce_kernel<<<>>> execution failed\n");
}
template void subap_reduce<float>(int size, int threads, int blocks, float *d_idata, float *d_odata);

template void subap_reduce<double>(int size, int threads, int blocks, double *d_idata, double *d_odata);


template <class T> void subap_reduce(int size, int threads, int blocks, T *d_idata, T *d_odata, T thresh)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps 
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
    reduce2<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, thresh, size);

    cutilCheckMsg("reduce_kernel<<<>>> execution failed\n");
}
template void subap_reduce<float>(int size, int threads, int blocks, float *d_idata, float *d_odata, float thresh);

template void subap_reduce<double>(int size, int threads, int blocks, double *d_idata, double *d_odata, double thresh);




template <class T> void subap_sortmax(int size, int threads, int blocks, T *d_idata, T *d_odata, int *values, int nmax)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps 
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * (sizeof(T) + sizeof(uint)) : threads * (sizeof(T) + sizeof(uint));
    sortmax<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, values, nmax);

    cutilCheckMsg("sortmax_kernel<<<>>> execution failed\n");
}
template void subap_sortmax<float>(int size, int threads, int blocks, float *d_idata, float *d_odata, int *values, int nmax);

template void subap_sortmax<double>(int size, int threads, int blocks, double *d_idata, double *d_odata, int *values, int nmax);


template <class T> void subap_centromax(int threads, int blocks, T *d_idata, T *d_odata, int npix, int nmax)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps 
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * (sizeof(T) + sizeof(uint)) : threads * (sizeof(T) + sizeof(uint));
    centroid_max<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata,npix, nmax,blocks);

    cutilCheckMsg("sortmax_kernel<<<>>> execution failed\n");
}
template void subap_centromax<float>( int threads, int blocks, float *d_idata, float *d_odata, int npix, int nmax);

template void subap_centromax<double>(int threads, int blocks, double *d_idata, double *d_odata, int npix, int nmax);




__global__ void subapnorm_krnl(float *odata, float *idata, float *fact, float *norm, float nphot,int n, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[tid] = idata[tid] * fact[tid/n] / norm[tid/n] * nphot;
    tid += blockDim.x * gridDim.x;
  }
}

int subap_norm(float *d_odata,float *d_idata,float *fact,float *norm,float nphot,int n, int N,int device)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  subapnorm_krnl<<<grid, threads>>>(d_odata, d_idata, fact, norm, nphot, n, N);

   return EXIT_SUCCESS;
}




template <class T> __global__ void reduce_phasex(T *g_idata, T *g_odata, T *offset, int *indx, unsigned int n, T alpha)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i   = blockIdx.x * n * n;
        
    sdata[tid] = (g_idata[indx[i+tid*n+n-1]]-offset[tid*n+n-1])-
      (g_idata[indx[i+tid*n]]-offset[tid*n]);
    
    __syncthreads();

    reduce_krnl(sdata,blockDim.x,tid);

    // write result for this block to global mem
    if (tid == 0) {
      g_odata[blockIdx.x] = sdata[0] / n * alpha;
    }
}

template <class T> __global__ void reduce_phasey(T *g_idata, T *g_odata, T *offset, int *indx, unsigned int n, T alpha)
// FIXME
// full parallelization would require to use 4 x threads
// instead of doing 4 operations per threads
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * n * n + threadIdx.x;
    
      sdata[tid] = (g_idata[indx[i+(n-1)*n]]-offset[(n-1)*n+tid])-
	(g_idata[indx[i]]-offset[tid]);

    __syncthreads();

    reduce_krnl(sdata,blockDim.x,tid);

    // write result for this block to global mem
    if (tid == 0) {
      g_odata[blockIdx.x] = sdata[0]/n*alpha;
    }
}

template <class T> void phase_reduce(int threads, int blocks, T *d_idata, T *d_odata, int *indx, T *offset, T alpha)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps 
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
    reduce_phasex<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, offset, indx, threads, alpha);

    cutilCheckMsg("reduce_phasex_kernel<<<>>> execution failed\n");

    reduce_phasey<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, &(d_odata[blocks]), offset, indx, threads, alpha);

    cutilCheckMsg("reduce_phasey_kernel<<<>>> execution failed\n");
}

template void phase_reduce<float>(int threads, int blocks, float *d_idata, float *d_odata, int *indx, float *offset, float alpha);

template void phase_reduce<double>(int threads, int blocks, double *d_idata, double *d_odata, int *indx, double *offset, double alpha);




template <class T> __global__ void derive_phasex(T *g_idata, T *g_odata, int *indx, T *offset, T *mask, T alpha, unsigned int n, unsigned int N, float *fluxPerSub)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (i < N) {
      if (tid % n == 0) {//start of a column
	sdata[tid] = ((g_idata[indx[i+1]]-offset[tid+1]) - 
		      (g_idata[indx[i]]-offset[tid])) * mask[indx[i]];
      } else {
	if ((tid+1) % n == 0) {//end of a column
	  sdata[tid] = ((g_idata[indx[i]]-offset[tid]) -
			(g_idata[indx[i-1]]-offset[tid-1])) * mask[indx[i]];
	} else 
	  sdata[tid] = ((g_idata[indx[i+1]]-offset[tid+1]) - 
			(g_idata[indx[i-1]]-offset[tid-1])) * mask[indx[i]];
      }
    }

    __syncthreads();

    reduce_krnl(sdata,blockDim.x,tid);

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0] / n / 2 * alpha / fluxPerSub[blockIdx.x];
}

template <class T> __global__ void derive_phasey(T *g_idata, T *g_odata, int *indx, T *offset, T *mask, T alpha, unsigned int n, unsigned int N, float *fluxPerSub)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (i < N) {
      if (tid < n) {//start of a column
	sdata[tid] = ((g_idata[indx[i+n]]-offset[tid+n]) - 
		      (g_idata[indx[i]]-offset[tid])) * mask[indx[i]];
      } else {
	if (tid >= n*(n-1)) {//end of a column
	  sdata[tid] = ((g_idata[indx[i]]-offset[tid]) -
			(g_idata[indx[i-n]]-offset[tid-n])) * mask[indx[i]];
	} else 
	  sdata[tid] = ((g_idata[indx[i+n]]-offset[tid+n]) - 
			(g_idata[indx[i-n]]-offset[tid-n])) * mask[indx[i]];
      }
    }

    __syncthreads();

    reduce_krnl(sdata,blockDim.x,tid);

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0] / n / 2 * alpha / fluxPerSub[blockIdx.x];
}

template <class T> void phase_derive(int size, int threads, int blocks, int n, T *d_idata, T *d_odata, int *indx, T *offset, T *mask, T alpha, float *fluxPerSub)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps 
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
    derive_phasex<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, indx, offset, mask, alpha, n, size,fluxPerSub);

    cutilCheckMsg("phase_derivex_kernel<<<>>> execution failed\n");

    derive_phasey<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, &(d_odata[blocks]), indx, offset, mask, alpha, n, size,fluxPerSub);

    cutilCheckMsg("phase_derivey_kernel<<<>>> execution failed\n");
 }

template void phase_derive<float>(int size, int threads, int blocks, int n, float *d_idata, float *d_odata, int *indx, float *offset, float *mask,float alpha, float *fluxPerSub);

template void phase_derive<double>(int size, int threads, int blocks, int n, double *d_idata, double *d_odata, int *indx, double *offset, double *mask,double alpha, float *fluxPerSub);



template <class T> __global__ void centroidx(T *g_idata, T *g_odata, T *alpha, unsigned int n, unsigned int N)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    sdata[tid] = (i < N) ? g_idata[i] * ((tid % n) +1) : 0;

    __syncthreads();

    reduce_krnl(sdata,blockDim.x,tid);

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0]*1.0/(alpha[blockIdx.x]+1.e-6);
}

template <class T> __global__ void centroidy(T *g_idata, T *g_odata, T *alpha, unsigned int n, unsigned int N)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    sdata[tid] = (i < N) ? g_idata[i] * ((tid / n) +1) : 0;

    __syncthreads();

    reduce_krnl(sdata,blockDim.x,tid);

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0]*1.0/(alpha[blockIdx.x]+1.e-6);
}

template <class T> void get_centroids(int size, int threads, int blocks, int n, T *d_idata, T *d_odata, T *alpha)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps 
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
    centroidx<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, alpha, n, size);

    cutilCheckMsg("centroidx_kernel<<<>>> execution failed\n");

    centroidy<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, &(d_odata[blocks]), alpha, n, size);

    cutilCheckMsg("centroidy_kernel<<<>>> execution failed\n");
}

template void get_centroids<float>(int size, int threads, int blocks, int n,float *d_idata, float *d_odata,float *alpha);

template void get_centroids<double>(int size, int threads, int blocks, int n,double *d_idata, double *d_odata, double *alpha);


template <class T> __global__ void centroidx(T *g_idata, T *g_odata, T *alpha, T thresh, unsigned int n, unsigned int N)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (i < N) sdata[tid] = (g_idata[i] > thresh) ? g_idata[i] * ((tid % n) +1) : 0;

    __syncthreads();

    reduce_krnl(sdata,blockDim.x,tid);

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0]*1.0/(alpha[blockIdx.x]+1.e-6);
}

template <class T> __global__ void centroidy(T *g_idata, T *g_odata, T *alpha, T thresh, unsigned int n, unsigned int N)
{
    T *sdata = SharedMemory<T>();

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (i < N) sdata[tid] = (g_idata[i] > thresh) ? g_idata[i] * ((tid / n) +1) : 0;

    __syncthreads();

    reduce_krnl(sdata,blockDim.x,tid);

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0]*1.0/(alpha[blockIdx.x]+1.e-6);
}

template <class T> void get_centroids(int size, int threads, int blocks, int n, T *d_idata, T *d_odata, T *alpha, T thresh)
{
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);

    // when there is only one warp per block, we need to allocate two warps 
    // worth of shared memory so that we don't index shared memory out of bounds
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
    centroidx<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, alpha, thresh, n, size);

    cutilCheckMsg("centroidx_kernel<<<>>> execution failed\n");

    centroidy<T><<< dimGrid, dimBlock, smemSize >>>(d_idata, &(d_odata[blocks]), alpha, thresh, n, size);

    cutilCheckMsg("centroidy_kernel<<<>>> execution failed\n");
}

template void get_centroids<float>(int size, int threads, int blocks, int n,float *d_idata, float *d_odata,float *alpha, float thresh);

template void get_centroids<double>(int size, int threads, int blocks, int n,double *d_idata, double *d_odata, double *alpha, double thresh);


template<class T> __global__ void roll_krnl(T *idata,int N,int  M,int Ntot)
{

  int tidt = threadIdx.x + blockIdx.x * blockDim.x;
  int nim  = tidt / Ntot;

  int tid = tidt - nim * Ntot;

  while (tid < Ntot) {
   
    
   int x=tid%N;
   int y=tid/N;

   int  xx=(x+N/2)%N;
   int  yy=(y+M/2)%M;
   int  tid2=xx+yy*N;

   __shared__ T tmp;
   tmp=idata[tid + nim * (N*M)];
   idata[tid + nim * (N*M)]=idata[tid2 + nim * (N*M)];
   idata[tid2 + nim * (N*M)]=tmp;

    tid += blockDim.x * gridDim.x;
  }
}

template<class T> int roll(T *idata,int N,int M,int nim)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, 0);
    
  long Ntot = N * M * nim;
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (Ntot/2 + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (Ntot/2 + nThreads -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);

  roll_krnl<<<grid, threads>>>(idata,N,M,Ntot/2);

  return EXIT_SUCCESS;

}

template int roll<float>(float *idata,int N,int M,int nim);

template int roll<double>(double *idata,int N,int M,int nim);

template int roll<hipFloatComplex>(hipFloatComplex *idata,int N,int M,int nim);


__global__ void fillcorr_krnl(hipFloatComplex *d_out, float *d_in,int npix_in,int npix_out,int N)
{
  int nim,npix,idx;
  int tid   = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    nim = tid / npix_in;
    npix = tid % npix_in;
    idx = nim * npix_out + npix;
    d_out[idx].x = d_in[tid];
    d_out[idx].y = 0.0;
    tid  += blockDim.x * gridDim.x;
  }
}

int fill_corr(hipFloatComplex *d_out, float *d_in, int npix_in, int npix_out, int N, int device)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
  
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);

  fillcorr_krnl<<<grid, threads>>>(d_out,d_in,npix_in,npix_out,N);
  cutilCheckMsg("fillcorr_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}

__global__ void fillval_krnl(hipFloatComplex *d_out, float val,int npix_in,int npix_out,int N)
{
  int nim,npix,idx;
  int tid   = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    nim = tid / npix_in;
    npix = tid % npix_in;
    idx = nim * npix_out + npix;
    d_out[idx].x = val;
    d_out[idx].y = 0.0;
    tid  += blockDim.x * gridDim.x;
  }
}

int fillval_corr(hipFloatComplex *d_out, float val, int npix_in, int npix_out, int N, int device)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
  
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);

  fillval_krnl<<<grid, threads>>>(d_out,val,npix_in,npix_out,N);
  cutilCheckMsg("fillcorr_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}


__global__ void abs2c_krnl(hipFloatComplex *odata, hipFloatComplex *idata, int N)
{
  __shared__ hipFloatComplex cache;
  int tid   = threadIdx.x + blockIdx.x * blockDim.x;


  if (tid < N) {
    cache = idata[tid];
    odata[tid].x = cache.x *cache.x +cache.y *cache.y;
    odata[tid].y = 0.0;
  } 
}


int abs2c(hipFloatComplex *d_odata, hipFloatComplex *d_idata, int N, int device)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
  
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);

  abs2c_krnl<<<grid, threads>>>(d_odata,d_idata,N);
  cutilCheckMsg("abs2c_kernel<<<>>> execution failed\n");

  return EXIT_SUCCESS;
}


__global__ void corr_krnl(hipFloatComplex *odata,hipFloatComplex *idata, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hipFloatComplex tmp;

  while (tid < N) {
    tmp.x = idata[tid].x*odata[tid].x+idata[tid].y*odata[tid].y;
    tmp.y = -1.0f*idata[tid].y*odata[tid].x+idata[tid].x*odata[tid].y;
    odata[tid] = tmp;
    tid += blockDim.x * gridDim.x;
  }
}

int correl(hipFloatComplex *d_odata,hipFloatComplex *d_idata,int N,int device)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  corr_krnl<<<grid, threads>>>(d_odata, d_idata, N);

   return EXIT_SUCCESS;
}

__global__ void corrnorm_krnl(float *odata,float *idata, int N)
{

  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < N) {
    odata[tid] /= (idata[tid]+1.e-6);
    tid += blockDim.x * gridDim.x;
  }
}

int corr_norm(float *d_odata,float *d_idata,int N,int device)
{

  struct hipDeviceProp_t deviceProperties;
  hipGetDeviceProperties(&deviceProperties, device);
    
  int maxThreads = deviceProperties.maxThreadsPerBlock;
  int nBlocks = deviceProperties.multiProcessorCount*8;
  int nThreads = (N + nBlocks -1)/nBlocks;

  if (nThreads > maxThreads) {
    nThreads = maxThreads;
    nBlocks = (N + nThreads  -1)/nThreads;
  }

  dim3 grid(nBlocks), threads(nThreads);
  //  dim3 grid(128), threads(128);

  corrnorm_krnl<<<grid, threads>>>(d_odata, d_idata, N);

   return EXIT_SUCCESS;
}
